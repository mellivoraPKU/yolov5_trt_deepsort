#include "hip/hip_runtime.h"
//
// Created by fagangjin on 25/3/2020.
//

#include "./ctnetPostprocess.h"
#include "basicOps.cuh"


namespace ctnet {


    __device__ float Logist(float data){ return 1./(1. + exp(-data)); }


    __global__ void CTdetforward_kernel(const float *hm, const float *reg,const float *wh ,
                                        float *output,const int w,const int h,const int classes,const int kernerl_size,const float visthresh  ) {
        int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
        if (idx >= w*h) return;
        int padding = kernerl_size/2;
        int offset = - padding /2;
        int stride = w*h;
        int grid_x = idx % w ;
        int grid_y = idx / w ;
        int cls,l,m;
        float c_x,c_y;
        for (cls = 0; cls < classes; ++cls )
        {
            int objIndex = stride * cls + idx;
            float objProb = hm[objIndex];
            float max=-1;
            int max_index =0;
            for(l=0 ;l < kernerl_size ; ++l)
                for(m=0 ; m < kernerl_size ; ++m){
                    int cur_x = offset + l + grid_x;
                    int cur_y = offset + m + grid_y;
                    int cur_index = cur_y * w + cur_x + stride*cls;
                    int valid = (cur_x>=0 && cur_x < w && cur_y >=0 && cur_y <h );
                    float val = (valid !=0 ) ? Logist(hm[cur_index]): -1;
                    max_index = (val > max) ? cur_index : max_index;
                    max = (val > max ) ?  val: max ;
                }
            objProb = Logist(objProb);
            if((max_index == objIndex) && (objProb > visthresh)){

                int resCount = (int)atomicAdd(output,1);
                //printf("%d",resCount);
                char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);
                Detection* det =  (Detection*)(data);
                c_x = grid_x + reg[idx] ; c_y  = grid_y + reg[idx+stride];
                det->bbox.x1 = (c_x - wh[idx]/2)*4;
                det->bbox.y1 = (c_y - wh[idx+stride]/2)*4 ;
                det->bbox.x2 = (c_x + wh[idx]/2)*4;
                det->bbox.y2 = (c_y + wh[idx+stride]/2)*4;
                det->classId = cls;
                det->prob = objProb;
            }
        }
    }

    __global__ void CTfaceforward_kernel(const float *hm, const float *wh,const float *reg,const float* landmarks,
                                         float *output,const int w,const int h,const int classes,const int kernerl_size,const float visthresh  ) {
        int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
        if (idx >= w*h) return;
        int padding = kernerl_size/2;
        int offset = - padding /2;
        int stride = w*h;
        int grid_x = idx % w ;
        int grid_y = idx / w ;
        int cls,l,m,mark_id;
        float c_x,c_y,scale_w,scale_h;
        for (cls = 0; cls < classes; ++cls )
        {
            int objIndex = stride * cls + idx;
            float objProb = hm[objIndex];
            float max=-1;
            int max_index =0;
            for(l=0 ;l < kernerl_size ; ++l)
                for(m=0 ; m < kernerl_size ; ++m){
                    int cur_x = offset + l + grid_x;
                    int cur_y = offset + m + grid_y;
                    int cur_index = cur_y * w + cur_x + stride*cls;
                    int valid = (cur_x>=0 && cur_x < w && cur_y >=0 && cur_y <h );
                    float val = (valid !=0 ) ? hm[cur_index]: -1;
                    max_index = (val > max) ? cur_index : max_index;
                    max = (val > max ) ?  val: max ;
                }
            //printf("%f\n",objProb);
            if((max_index == objIndex) && (objProb > visthresh)){

                int resCount = (int)atomicAdd(output,1);
                //printf("%d",resCount);
                char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);
                Detection* det =  (Detection*)(data);
                c_x = (grid_x + reg[idx+stride] + 0.5)*4 ; c_y  = (grid_y + reg[idx] + 0.5) * 4;
                scale_w =  expf(wh[idx+stride]) * 4 ; scale_h  = expf(wh[idx]) * 4;
                det->bbox.x1 = c_x - scale_w/2;
                det->bbox.y1 = c_y - scale_h/2 ;
                det->bbox.x2 = c_x + scale_w/2;
                det->bbox.y2 = c_y + scale_h/2;
                det->prob = objProb;
                det->classId = cls;
                for(mark_id=0 ; mark_id < 5 ; ++mark_id ){
                    det->marks[mark_id].x = det->bbox.x1 + landmarks[idx + (2*mark_id+1)*stride]*scale_w;
                    det->marks[mark_id].y = det->bbox.y1 + landmarks[idx + (2*mark_id)*stride]*scale_h;
                }
            }
        }
    }

    void CTdetforward_gpu(const float *hm, const float *reg,const float *wh ,float *output,
                          const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
        uint num = w * h;
        CTdetforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,reg,wh,output,w,h,classes,kernerl_size,visthresh);
    }

    void CTfaceforward_gpu(const float *hm, const float *wh,const float *reg,const float* landmarks,float *output,
                           const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
        uint num = w * h;
        CTfaceforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,wh,reg,landmarks,output,w,h,classes,kernerl_size,visthresh);
    }

}